#include "hip/hip_runtime.h"
#include <sstream>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>


static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
    if(err!=hipSuccess)
    {
        fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",msg,file_name,line_number,hipGetErrorString(err));
        std::cin.get();
        exit(EXIT_FAILURE);                                
    }
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call), (msg), __FILE__, __LINE__)

/*
 Main function to convert panorama image to perspective image
 */
__global__ 
void pano2perspective(
    unsigned char *pano,
    unsigned char *pers,
    double *im2ori,
    int pano_w, int pano_h,
    int pers_w, int pers_h,
    int pano_step, int pers_step)
{
    // 2D Index of current thread
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Only valid threads perform memory I/O 
	if((i < pers_w) && (j < pers_h))
    {
        // Location of colored pixel in output
        int pers_tid = j * pers_step + (3 * i);
        // Create orientation matrix
        double ori[3];
        for(int m=0; m<3; m++)
        {
            ori[m] = im2ori[m*3] * (double)i
                + im2ori[m*3+1] * (double)j
                + im2ori[m*3+2]; 
        }
        
        double D = sqrt(
            ori[0] * ori[0]
            + ori[1] * ori[1]
            + ori[2] * ori[2]);
        
        double phi = asin(ori[1] / D); // [-pi/2:pi/2]
        double theta = atan2(ori[0], ori[2]); // [-pi:pi]
        double u = (theta + M_PI) * (pano_w/(2.0*M_PI)) - .5;
        double v = (phi + M_PI/2) * (pano_h/(M_PI)) - .5;
        int px0 = (int)v*pano_step + 3*(int)u;
        int px1 = (int)(v+1)*pano_step + 3*(int)u;
        int px2 = (int)v*pano_step + 3*(int)(u+1);
        int px3 = (int)(v+1)*pano_step + 3*(int)(u+1);
        double v0 = (1.0-(v-(int)v)) * (1.0-(u-(int)u));
        double v1 = ((v-(int)v))*(1.0 - (u-(int)u));
        double v2 = (1.0 - (v-(int)v))*((u-(int)u));
        double v3 = ((v-(int)v))*((u-(int)u));
        pers[pers_tid] = static_cast<unsigned char>(
            pano[px0] * v0
            + pano[px1] * v1
            + pano[px2] * v2
            + pano[px3] * v3
        );
        pers[pers_tid + 1] = static_cast<unsigned char>(
            pano[px0 + 1] * v0
            + pano[px1 + 1] * v1
            + pano[px2 + 1] * v2
            + pano[px3 + 1] * v3
        );
        pers[pers_tid + 2] = static_cast<unsigned char>(
            pano[px0 + 2] * v0
            + pano[px1 + 2] * v1
            + pano[px2 + 2] * v2
            + pano[px3 + 2] * v3
        );
    }
}

void process_image(
    const cv::Mat &pano,
    cv::Mat &pers,
    const cv::Mat_<double> &rot,
    const cv::Mat_<double> &K)
{
    // preprocess
	cv::Mat T_im2ori = rot.inv() * K.inv();
    
    // Calculate total number of bytes of input image and orientation matrix
    const int panoBytes = pano.step * pano.rows;
    const int persBytes = pers.step * pers.rows;
    const int doubleBytes = T_im2ori.step * T_im2ori.rows;

    // Return pointers
    unsigned char *d_pano, *d_pers;
    double *im2ori = new double[9];
    double *d_im2ori;
       
    // Allocate device memory
    SAFE_CALL(hipMalloc<unsigned char>(&d_pano, panoBytes),"CUDA Malloc Failed");
    SAFE_CALL(hipMalloc<unsigned char>(&d_pers, persBytes),"CUDA Malloc Failed");
    SAFE_CALL(hipMalloc((void **)&d_im2ori, 9*sizeof(double)),"CUDA Malloc Failed");

    for(int i=0; i<3; i++)
    {
        im2ori[i*3] = T_im2ori.at<double>(i, 0);
        im2ori[i*3 + 1] = T_im2ori.at<double>(i, 1);
        im2ori[i*3 + 2] = T_im2ori.at<double>(i, 2);
    }

    //Copy data from OpenCV input image to device memory
    SAFE_CALL(hipMemcpy(d_pano, pano.ptr(), panoBytes, hipMemcpyHostToDevice),"CUDA Memcpy Host To Device Failed");
    SAFE_CALL(hipMemcpy(d_im2ori, im2ori, 9*sizeof(double), hipMemcpyHostToDevice),"CUDA Memcpy Host To Device Failed");
    
    // Specify device
    // int device = 0;
    // hipGetDevice(&device);

    // Specify a reasonable block size
    const dim3 block(16, 16);
    //Calculate grid size to cover the whole image
    const dim3 grid((pers.cols + block.x - 1)/block.x, (pers.rows + block.y - 1)/block.y);
                    
    // Launch the color conversion kernel
    // std::cout << pano.step << " " << pano.cols << " " << pano.rows << std::endl;
    pano2perspective<<<grid, block>>>(d_pano, d_pers, d_im2ori,
                                    pano.cols, pano.rows,
                                    pers.cols, pers.rows, 
                                    pano.step, pers.step);
    
    //Synchronize to check for any kernel launch errors
    SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed");
    
    //Copy back data from destination device meory to OpenCV output image
    SAFE_CALL(hipMemcpy(pers.ptr(), d_pers, persBytes, hipMemcpyDeviceToHost),"CUDA Memcpy Host To Device Failed");
    
    //Free the device memory
    SAFE_CALL(hipFree(d_pano),"CUDA Free Failed");
    SAFE_CALL(hipFree(d_pers),"CUDA Free Failed");
    SAFE_CALL(hipFree(d_im2ori),"CUDA Free Failed");

}
